/**
 * @file
 * @author Wayne Madden <s3197676@student.rmit.edu.au>
 * @version 0.3
 *
 * @section LICENSE
 * Free to re-use and reference from within code as long as the original owner
 * is referenced as per GNU standards
 *
 * @section DESCRIPTION
 * Basic matrix solver. Accepts two input matrices A and B in the form of
 * equation 'A . X = B'. Uses gaussian elimination.
 */

#include "SolverMain.h"

/**
 * Main method of matrix solver program
 *
 * @param argc Number of items to be stored
 * @param argsv[] Size to be used for each item by the Memory Manager
 */
int main(int argc, char* argv[])
{
   float *hA, *dA;
   float *hX, *dX;
   float *hB, *dB;
   int i, j;
   int matrixSide;
   int status;
   FILE* fp;
   char line[BUFFER_SIZE];
   char* token;
   //LARGE_INTEGER start, end, freq;
   int n, gridCount, blockCount, threads;

   /*validate arguments*/
   if (argc != 4)
   {
      cout << "Invalid arguments! Press enter to continue...";
      cin.ignore(1);
	  exit(EXIT_FAILURE);
   }

   /*necesary for microsecond accuracy*/
   //QueryPerformanceFrequency(&freq);

   /*1. init*/
   //QueryPerformanceCounter(&start);
   /*set device*/
   status = hipSetDevice(0);
   if (status != hipSuccess)
   {
      cout << "No valid device found! Press enter to continue...";
      cin.ignore(1);
	  exit(EXIT_FAILURE);
   }

   /*read command line input*/
   matrixSide = atoi(argv[1]);

   /*allocate host memory*/
   hA = new float[matrixSide * matrixSide];
   hX = new float[matrixSide];
   hB = new float[matrixSide];

   /*check host memory*/
   if (hA == NULL || hX == NULL || hB == NULL)
   {
      cout << "Unable to allocate host memory! Press enter to continue...";
      cin.ignore(1);
	  exit(EXIT_FAILURE);
   }

   /*allocate device memory*/
   status = hipSuccess;
   status += hipMalloc((void**) &dA, sizeof(float) * matrixSide * matrixSide);
   status += hipMalloc((void**) &dX, sizeof(float) * matrixSide);
   status += hipMalloc((void**) &dB, sizeof(float) * matrixSide);

   /*check device memory*/
   if (status != hipSuccess)
   {
      cout << "Unable to allocate host memory! Press enter to continue...";
      cin.ignore(1);
	  exit(EXIT_FAILURE);
   }

   /*read matrix a into memory*/
   fp = fopen(argv[2], "r");
   i = 0;
   while (i < matrixSide && fgets(line, 1000000, fp) != NULL)
   {
      j = 0;
      token = strtok(line, " ,");
      do
      {
         hA[i * matrixSide + j] = atof(token);
         j++;
      } while (j < matrixSide && (token = strtok(NULL, " ,")) != NULL);
      i++;
   }
   fclose(fp);

   /*read matrix b into memory*/
   fp = fopen(argv[3], "r");
   i = 0;
   while (i < matrixSide && fgets(line, 1000, fp) != NULL)
   {
      hB[i] = atof(line);
      i++;
   }
   fclose(fp);

   /*copy host memory to device*/
   hipMemcpy(dA, hA, sizeof(float) * matrixSide * matrixSide,
                                                      hipMemcpyHostToDevice);
   hipMemcpy(dB, hB, sizeof(float) * matrixSide, hipMemcpyHostToDevice);

   //QueryPerformanceCounter(&end);
   //cout << (double)(end.QuadPart - start.QuadPart) / freq.QuadPart << ",";

   /*first display*/
   displayEquation(hA, NULL, hB, matrixSide);

   /*calculate threads per block per grid. blockDim is optimally a multiple of
   32 (warp size) to keep sm busy (better, 64 as two warps can be interleaved)*/
   n = 2;
   threads = (matrixSide / (WARP_SIZE * n) + 1) * (WARP_SIZE * n);
   do
   {
      gridCount = threads / (WARP_SIZE * n);
      blockCount = threads / gridCount;
      n += 2;
   } while (gridCount > GRID_MAX);
   if (blockCount >= 768)
   {
      cout << "Matrix is too large for this solver! Press enter to continue...";
      cin.ignore(1);
      exit(EXIT_FAILURE);
   }

   /*2. solve matrix*/
   //QueryPerformanceCounter(&start);
   gaussianElimination(dA, dX, dB, matrixSide, gridCount, blockCount);
   //QueryPerformanceCounter(&end);
   //cout << (double)(end.QuadPart - start.QuadPart) / freq.QuadPart << ",";

   /*copy device memory to host*/
   hipMemcpy(hX, dX, sizeof(float) * matrixSide, hipMemcpyDeviceToHost);

   /*display the solved values of matrix X*/
   displayEquation(hA, hX, hB, matrixSide);
   
   /*4. cleanup*/
   //QueryPerformanceCounter(&start);
   /*free host memory*/
   delete[] hA;
   delete[] hX;
   delete[] hB;

   /*free device memory*/
   hipFree(dA);
   hipFree(dX);
   hipFree(dB);

   /*reset device for profiling tool traces*/
   hipDeviceReset();

   //QueryPerformanceCounter(&end);
   //cout << (double)(end.QuadPart - start.QuadPart) / freq.QuadPart;

   /*prompt to continue - to allow the user to read output before exiting*/
   cout << "Press enter to continue...";
   cin.ignore(1);

   exit(EXIT_SUCCESS);
}
